
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__global__ void vAdd(int* a, int* b, int* c, int n){
	int i = threadIdx.x;
	if(i<n)
		c[i] = a[i] + b[i];
}

void wrapper(int* a, int* b, int* c, int n){
	
	int *d_a,*d_b,*d_c;

	hipMalloc(&d_a,n*sizeof(int));
	hipMalloc(&d_b,n*sizeof(int));
	hipMalloc(&d_c,n*sizeof(int));

	hipMemcpy(d_a,a,n * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,n * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_c,c,n * sizeof(int),hipMemcpyHostToDevice);

	clock_t start = clock();
	vAdd<<< 1,n >>>(d_a,d_b,d_c,n);
	clock_t end = clock();
	printf("Took %f Seconds", float(end-start)/CLOCKS_PER_SEC);

	//cudaMemcpy(a,d_a,n*sizeof(int),cudaMemcpyDeviceToHost);
	//cudaMemcpy(b,d_b,n*sizeof(int),cudaMemcpyDeviceToHost);
	hipMemcpy(c,d_c,n*sizeof(int),hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return;
}
